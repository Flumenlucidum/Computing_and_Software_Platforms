#include "hip/hip_runtime.h"
#include<math.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<R.h>
#include<omp.h>
static double* gpu_mem_X;
static double* gpu_mem_W;
static double* gpu_mem_B;
static double* gpu_mem_Result;

__global__ void matmul(double* A, double* B, double* C, int M, int N, int K) 
{
	  int i = blockDim.x * blockIdx.x + threadIdx.x;
	  int j = blockDim.y * blockIdx.y + threadIdx.y;
	  if (i >= M || j >= N) return;
	  double  s=0;
	  #pragma omp parallel for 
	  for (int k = 0; k < K; ++k)
	  {
	 	s += A[i * K + k] * B[k * N + j];
	  }
	  C[i*N+j]=s;
}

__global__ void sqr_cos(double* wx, double* bias,int num_f, int row)
{
 	  int i = blockDim.x * blockIdx.x + threadIdx.x;
	  int j = blockDim.y * blockIdx.y + threadIdx.y;
	  if (i >= num_f || j >= row) return;
	  
	  double s=bias[i];
	  double w=wx[i*row+j];
	  wx[i*row+j]=sqrt((float)2)*cos(w+s);

}	

extern "C" void rand_four(double* weight, double* x_transposed, double*bias, double*result, int* num_f_p, int* col_length_p,int* row_p) 
{	
	int num_f=*num_f_p, col_length=*col_length_p, row=*row_p;
	//Memory allocation
	hipMalloc(&gpu_mem_X, col_length*row* sizeof(double));
	hipMalloc(&gpu_mem_W, num_f*col_length*sizeof(double));
	hipMalloc(&gpu_mem_Result,num_f*row*sizeof(double));
	hipMalloc(&gpu_mem_B,num_f*sizeof(double));
	hipDeviceSynchronize();
	//Send the object from host to GPU
	hipMemcpy(gpu_mem_W, weight, num_f *col_length * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_mem_X, x_transposed, col_length*row* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_mem_B, bias, col_length* sizeof(double), hipMemcpyHostToDevice);
	
	
	dim3 blockDim(5,61,1);
	dim3 gridDim(num_f/5,313*18,1);
	matmul<<<gridDim,blockDim>>>(gpu_mem_W,gpu_mem_X,gpu_mem_Result,num_f,row,col_length);
	hipDeviceSynchronize();
	sqr_cos<<<gridDim,blockDim>>>(gpu_mem_Result, gpu_mem_B, num_f, row);
	hipDeviceSynchronize();
	//Send the result from GPU to host
	hipMemcpy(result, gpu_mem_Result, num_f * row * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(gpu_mem_X);
	hipFree(gpu_mem_W);
	hipFree(gpu_mem_B);
	hipFree(gpu_mem_Result);
}

extern "C" void mat_mul(double* A, double*B, double* C, int* M_p, int* N_p,int* K_p) 
{	

	int M=*M_p, N=*N_p, K=*K_p;
	hipMalloc(&gpu_mem_X, M*K* sizeof(double));
	hipMalloc(&gpu_mem_W, K*N*sizeof(double));
	hipMalloc(&gpu_mem_Result,M*N*sizeof(double));
	//hipDeviceSynchronize();

	hipMemcpy(gpu_mem_X, A,M*K* sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(gpu_mem_W, B, K*N* sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy(gpu_mem_Result, C, M*N* sizeof(double), hipMemcpyHostToDevice);
	
	dim3 blockDim(61,5,1);
	dim3 gridDim(313*18,1,1);
	matmul<<<gridDim,blockDim>>>(gpu_mem_X,gpu_mem_W,gpu_mem_Result,M,N,K);
	//hipDeviceSynchronize();
	hipMemcpy(C, gpu_mem_Result, M*N* sizeof(double), hipMemcpyDeviceToHost);
	hipFree(gpu_mem_X);
	hipFree(gpu_mem_W);
	hipFree(gpu_mem_Result);
}



